#include "hip/hip_runtime.h"
#include ""
#include "../high_performance_timer/High_performance_timer.h" 

#include <iostream>
#include <string>
#include <stdlib.h>
#include <time.h>
#include<omp.h>


using namespace std;

bool mem_alloc(int **a, int **b, int **c, int size);
void clean_up(int *a, int *b, int *c);
void fill_arrays(int *a, int *b, int *c, int size);
void add_vec_serial_CPU(int * a, int * b, int * c, int size);

hipError_t add_array_cuda(int *c, const int *a, const int, *b, unsigned int size);



int main(int argc, char * argv[]) {
	//seed random with time
	srand((unsigned)time(NULL));

	//set up the timer
	HighPrecisionTime htp;

	//set the second argument to be 1000 by default
	int size = 1000;
	//set the iterations to be 100 by default
	int iter = 100;

	//declare the variables
	int *a = nullptr;
	int *b = nullptr;
	int *c = nullptr;

	//check to see the user added a second argument
	//if they did, then change the size to that new argument
	if (argc > 1) {
		size = stoi(argv[1]);
	}
	cout << "the size of the array is: " << size << endl;

	//check to see if the user added a third argument
	//if they did, then change the size to that new argument
	if (argc > 2) {
		iter = stoi(argv[2]);
	}
	cout << "the number of iterations is: " << iter << endl;

	//try to allocate the memory
	try {
		if (!mem_alloc(&a, &b, &c, size)) {
			throw("did not correctly allocate!");
		}
		cout << "memory has been allocated!" << endl;
	}
	//if it doesn't work print out the error message and continue
	//to clean up
	catch(char * err_message) {
		cout << err_message << endl;
	}

	fill_arrays(a, b, c, size);

	double t = 0;


	

	htp.TimeSinceLastCall();
	for (int i = 0; i < iter; i++) {
		add_vec_serial_CPU(a, b, c, size);
		t = t + htp.TimeSinceLastCall();
	}
		
	t = t / iter;
	cout << "Add vec serial CPU took:   " << t << "  seconds!" << endl;





	
	clean_up(a, b, c);
	return 0;
}



//---------------------------------------------------------------------------
//function to allocate memory
bool mem_alloc(int **a, int **b, int **c, int size) {
	//set up the return value to be false
	bool retval = false;

	//allocate memory for all the arrays and size
	*a = (int *)malloc(sizeof(int) * size);
	*b = (int *)malloc(sizeof(int) * size);
	*c = (int *)malloc(sizeof(int) * size);

	//check to make sure they properly allocated
	//if they were then change retval to true
	if (*a != NULL || *b != NULL || *c != NULL) {
		retval = true;
	}

	return retval;
}

//---------------------------------------------------------------------------
//function for cleaning up and freeing the data
void clean_up(int *a, int *b, int *c) {
	free(a);
	free(b);
	free(c);

	if (a != nullptr) {
		a = nullptr;
	}
	if (a != nullptr) {
		b = nullptr;
	}
	if (a != nullptr) {
		c = nullptr;
	}
}

//---------------------------------------------------------------------------
void fill_arrays(int *a, int *b, int *c, int size) {
	//fill in the arrays 
	for (int i = 0; i < size; i++) {
		a[i] = rand() % 20 + 1;
		b[i] = rand() % 20 + 1;
		c[i] = 0;
	}
}




//---------------------------------------------------------------------------
void add_vec_serial_CPU(int * a, int * b, int * c, int size) {
	//add a and b and save it into c
	for (int i = 0; i < size; i++) {
		c[i] = a[i] + b[i];
	}
}
