#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "../high_performance_timer/High_performance_timer.h" 

#include <iostream>
#include <string>
#include <stdlib.h>
#include <time.h>
#include<omp.h>


using namespace std;

bool mem_alloc(int **a, int **b, int **c, int size);
void clean_up(int *a, int *b, int *c);
void fill_arrays(int *a, int *b, int *c, int size);
void add_vec_serial_CPU(int * a, int * b, int * c, int size);

void cuda_malloc_add(int* cpu_a, int* cpu_b, int* cpu_c, int size);


__global__ void add_kernel(int *c, const int *a, const int *b, int size) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] + b[i];
	}
}



int main(int argc, char * argv[]) {
	//seed random with time
	srand((unsigned)time(NULL));

	//set up the timer
	HighPrecisionTime htp;

	//set the second argument to be 1000 by default
	int size = 1000;
	//set the iterations to be 100 by default
	int iter = 100;

	//declare the variables
	int *a = nullptr;
	int *b = nullptr;
	int *c = nullptr;

	//check to see the user added a second argument
	//if they did, then change the size to that new argument
	if (argc > 1) {
		size = stoi(argv[1]);
	}
	cout << "the size of the array is: " << size << endl;

	//check to see if the user added a third argument
	//if they did, then change the size to that new argument
	if (argc > 2) {
		iter = stoi(argv[2]);
	}
	cout << "the number of iterations is: " << iter << endl;

	//try to allocate the memory
	try {
		if (!mem_alloc(&a, &b, &c, size)) {
			throw("did not correctly allocate!");
		}
		cout << "memory has been allocated!" << endl;
	}
	//if it doesn't work print out the error message and continue
	//to clean up
	catch(char * err_message) {
		cout << err_message << endl;
	}

	fill_arrays(a, b, c, size);

	double t = 0;


	

	htp.TimeSinceLastCall();
	for (int i = 0; i < iter; i++) {
		add_vec_serial_CPU(a, b, c, size);
		t = t + htp.TimeSinceLastCall();
	}
	cout << "\ntotal time to add vec serial CPU took: " << t << " seconds!" << endl;
	t = t / iter;
	cout << "Add vec serial CPU took:   " << t << "  seconds!" << endl;





	//=====================test cuda code=============================
	//cuda_malloc_add(a, b, c, size);
	hipError_t cuda_status;

	int * gpu_a = nullptr;
	int * gpu_b = nullptr;
	int * gpu_c = nullptr;

	

	try {
		//choose which GPU to run on, change this on a multi-GPU system.
		cuda_status = hipSetDevice(0);
		if (cuda_status != hipSuccess) {
			throw("cudaSetDeice failed!");
		}

		//allocate GPU buffers for 3 arrays 
		cuda_status = hipMalloc((void**)&gpu_a, size * sizeof(int));
		if (cuda_status != hipSuccess) {
			throw("hipMalloc of array a failed!");
		}
		cuda_status = hipMalloc((void**)&gpu_b, size * sizeof(int));
		if (cuda_status != hipSuccess) {
			throw("hipMalloc of array b failed!");
		}
		cuda_status = hipMalloc((void**)&gpu_c, size * sizeof(int));
		if (cuda_status != hipSuccess) {
			throw("hipMalloc of array c failed!");
		}


		//copy the vectors over to the GPU buffers
		//only copy over a & b cause they are the only ones with any real data
		htp.TimeSinceLastCall();
		cuda_status = hipMemcpy(gpu_a, a, size * sizeof(int), hipMemcpyHostToDevice);
		if (cuda_status != hipSuccess) {
			throw("hipMemcpy of array a failed!");
		}
		cuda_status = hipMemcpy(gpu_b, b, size * sizeof(int), hipMemcpyHostToDevice);
		if (cuda_status != hipSuccess) {
			throw("hipMemcpy of array a failed!");
		}
		t =htp.TimeSinceLastCall();
		cout << "\ncuda mem copy took: " << t << "seconds!" << endl;
	}
	catch (char * err_message) {
		cout << err_message << endl;
		goto Error;
	}




	//re-initialize t to be 0 for the new timing
	t = 0;
	//now add the vectors together
	
	for (int i = 0; i < iter; i++) {
		htp.TimeSinceLastCall();
		add_kernel <<<1, size >>> (gpu_c, gpu_a, gpu_b, size);
		t = t + htp.TimeSinceLastCall();
	}


	cout << "cuda add arrays total took: " << t << " seconds!" << endl;

	t = t / iter;

	cout << "cuda add arrays took: " << t << " seconds!" << endl;



Error:
	hipFree(gpu_c);
	hipFree(gpu_b);
	hipFree(gpu_a);
	








	
	clean_up(a, b, c);
	return 0;
}



//---------------------------------------------------------------------------
//function to allocate memory
bool mem_alloc(int **a, int **b, int **c, int size) {
	//set up the return value to be false
	bool retval = false;

	//allocate memory for all the arrays and size
	*a = (int *)malloc(sizeof(int) * size);
	*b = (int *)malloc(sizeof(int) * size);
	*c = (int *)malloc(sizeof(int) * size);

	//check to make sure they properly allocated
	//if they were then change retval to true
	if (*a != NULL || *b != NULL || *c != NULL) {
		retval = true;
	}

	return retval;
}

//---------------------------------------------------------------------------
//function for cleaning up and freeing the data
void clean_up(int *a, int *b, int *c) {
	free(a);
	free(b);
	free(c);

	if (a != nullptr) {
		a = nullptr;
	}
	if (a != nullptr) {
		b = nullptr;
	}
	if (a != nullptr) {
		c = nullptr;
	}
}

//--------------------------------------------------------------------------- 
void fill_arrays(int *a, int *b, int *c, int size) {
	//fill in the arrays 
	for (int i = 0; i < size; i++) {
		a[i] = rand() % 20 + 1;
		b[i] = rand() % 20 + 1;
		c[i] = 0;
	}
}




//---------------------------------------------------------------------------
void add_vec_serial_CPU(int * a, int * b, int * c, int size) {
	//add a and b and save it into c
	for (int i = 0; i < size; i++) {
		c[i] = a[i] + b[i];
	}
}


//=========CUDA CODE===================
//---------------------------------------------------------------------------
void cuda_malloc_add(int * cpu_a, int * cpu_b, int * cpu_c, int size) {
	hipError_t cuda_status;

	int * gpu_a = nullptr;
	int * gpu_b = nullptr;
	int * gpu_c = nullptr;

	int malloc_size = size * sizeof(int);

	try {
		//choose which GPU to run on, change this on a multi-GPU system.
		cuda_status = hipSetDevice(0);
		if (cuda_status != hipSuccess) {
			throw("cudaSetDeice failed!");
		}

		//allocate GPU buffers for 3 arrays 
		cuda_status = hipMalloc((void**)&gpu_a, malloc_size);
		if (cuda_status != hipSuccess) {
			throw("hipMalloc of array a failed!");
		}
		cuda_status = hipMalloc((void**)&gpu_b, malloc_size);
		if (cuda_status != hipSuccess) {
			throw("hipMalloc of array b failed!");
		}
		cuda_status = hipMalloc((void**)&gpu_c, malloc_size);
		if (cuda_status != hipSuccess) {
			throw("hipMalloc of array c failed!");
		}


		//copy the vectors over to the GPU buffers
		//only copy over a & b cause they are the only ones with any real data
		cuda_status = hipMemcpy(gpu_a, cpu_a, malloc_size, hipMemcpyHostToDevice);
		if (cuda_status != hipSuccess) {
			throw("hipMemcpy of array a failed!");
		}
		cuda_status = hipMemcpy(gpu_b, cpu_b, malloc_size, hipMemcpyHostToDevice);
		if (cuda_status != hipSuccess) {
			throw("hipMemcpy of array a failed!");
		}
	}
	catch (char * err_message) {
		cout << err_message << endl;
		goto Error;
	}


	//now add the vectors together
	add_kernel <<<1, size >> > (gpu_c, gpu_a, gpu_b, size);
	cout << gpu_a[0] << endl;



Error:
	hipFree(gpu_c);
	hipFree(gpu_b);
	hipFree(gpu_a);
}


